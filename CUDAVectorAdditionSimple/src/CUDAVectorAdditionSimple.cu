#include <iostream>
#include <hip/hip_runtime.h>
using std::cout;
using std::endl;

#define TYPE float
#define BLOCKSIZE 32	// Workgroup size

// kernel
__global__ void add_kernel(TYPE* A, TYPE* B, TYPE* C) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    C[n] = A[n]+B[n];
}

int main()
{
	// Sizes
	const unsigned int N = 1024u;
	dim3 block(BLOCKSIZE);		// Workgroup size
	dim3 grid(N/BLOCKSIZE);		// No. of workgroups = Global threads/workgroup size

	// Host and Device pointers;
	TYPE *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Host memory allocation
	h_A = new TYPE[N];
	h_B = new TYPE[N];
	h_C = new TYPE[N];

	// host data initialisation
    for (unsigned int i = 0; i<N; i++)
    {
		h_A[i] = 3.f;
		h_B[i] = 2.f;
		h_C[i] = 1.f;
    }

	// Allocate device memory
    hipMalloc((void**)&d_A, N*sizeof(TYPE));
    hipMalloc((void**)&d_B, N*sizeof(TYPE));
    hipMalloc((void**)&d_C, N*sizeof(TYPE));

    // Copy host A and B arrays to GPU
    hipMemcpy(d_A, h_A, N*sizeof(TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N*sizeof(TYPE), hipMemcpyHostToDevice);

	cout << "Value of C[0] before kernel launch: " << h_C[0] << endl;

    // Launch kernel
    add_kernel <<<grid, block>>> (d_A, d_B, d_C);

	// Copy device C to host C
    hipMemcpy(h_C, d_C, N*sizeof(TYPE), hipMemcpyDeviceToHost);

	cout << "Value of C[0] after kernel execution: " << h_C[0] << endl;

    // Cleanup
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
